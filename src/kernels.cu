#include <stdint.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

hipblasHandle_t cublas_handle;

__global__ void dequantize(float *out, int8_t *quants, float *scales, uint16_t group_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = quants[i] * scales[i / group_size];
}

extern "C"
{
    void cuda_init()
    {
        hipblasCreate(&cublas_handle);
    }

    void cuda_matmul(float *out, int8_t *a_h_quants, int8_t *b_h_quants, float *a_h_scales, float *b_h_scales, uint32_t a, uint32_t b, uint32_t n, uint16_t group_size)
    {
        int8_t *a_d_quants, *b_d_quants;
        float *out_d_values, *a_d_values, *b_d_values, *a_d_scales, *b_d_scales;
        const uint32_t a_len = a * n, b_len = b * n, out_len = a * b;
        hipMalloc(&out_d_values, out_len * sizeof(float));
        hipMalloc(&a_d_values, a_len * sizeof(float));
        hipMalloc(&b_d_values, b_len * sizeof(float));
        hipMalloc(&a_d_quants, a_len * sizeof(int8_t));
        hipMalloc(&b_d_quants, b_len * sizeof(int8_t));
        hipMalloc(&a_d_scales, a_len / group_size * sizeof(float));
        hipMalloc(&b_d_scales, b_len / group_size * sizeof(float));
        hipMemcpy(a_d_quants, a_h_quants, a_len * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMemcpy(b_d_quants, b_h_quants, b_len * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMemcpy(a_d_scales, a_h_scales, a_len / group_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(b_d_scales, b_h_scales, b_len / group_size * sizeof(float), hipMemcpyHostToDevice);
        dequantize<<<a_len / group_size, group_size>>>(a_d_values, a_d_quants, a_d_scales, group_size);
        dequantize<<<b_len / group_size, group_size>>>(b_d_values, b_d_quants, b_d_scales, group_size);
        float alpha = 1, beta = 0;
        hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, a, b, n, &alpha, a_d_values, n, b_d_values, n, &beta, out_d_values, a);
        hipMemcpy(out, out_d_values, out_len * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(out_d_values);
        hipFree(a_d_values);
        hipFree(b_d_values);
        hipFree(a_d_quants);
        hipFree(b_d_quants);
        hipFree(a_d_scales);
        hipFree(b_d_scales);
    }
}
